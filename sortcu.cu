#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include "hip/hip_runtime.h"
#include ""
#include "sortcu.h"
#define MOD_MAX_MY 4294967295


#define MOD_MAX_SUM_MY(a,b) ( (long(a)+long(b)) % MOD_MAX_MY )

__global__ void sortcu( uint32_t *data_arr,uint32_t *prefix_arr, int ndata);
__global__ void prefix_up(uint32_t *B_this, uint32_t *B_prev, int num_ele_this);
__global__ void prefix_down(uint32_t *B_h, uint32_t *C_h, uint32_t *C_hp1, int num_ele_this) ;
__global__ void init_idx_arr(uint32_t *index_arr,int num_ele_this);
__global__ void d_idx_convert(uint32_t *index_arr,uint32_t * d_prfx ,int num_ele_this);
__device__ int64_t d_step;
__device__ int64_t d_substep;
using namespace std;
// check cuda error
void check_cuda_error(hipError_t err, int line) {
  if (err != hipError_t::hipSuccess) {
    std::cerr << "CUDA error at line " << line << " : "
              << hipGetErrorString(err) << "\n";
    std::exit(1);
  }
}

#define CHECK_ERROR(err) check_cuda_error(err, __LINE__)

// sets up and calls GPU kernel
void sort(uint32_t *data, int ndata) {
  uint32_t *h_data = data; //host pointer
  const int num_data = ndata;
  const long num_bytes = num_data * sizeof(uint32_t);

  int least_pow2 = 0;
  while ((1ULL << least_pow2) < num_data) {
    least_pow2++;
  }
  
  const int padded_num_data = (1 << least_pow2); //number of elements after padding
  const long padded_num_bytes = padded_num_data * sizeof(uint32_t); // number of bytes with padding
  const int pad_num_data = padded_num_data - num_data; // number of elements to be padded
 
  uint32_t **B = new uint32_t*[least_pow2+1];
  uint32_t **C = new uint32_t*[least_pow2+1];
  uint32_t *this_addr=nullptr;
  
  CHECK_ERROR(hipMalloc(&this_addr, padded_num_bytes));
  B[0]=this_addr;
  
  CHECK_ERROR(hipMemcpy(B[0] + pad_num_data, h_data, num_bytes,
                    hipMemcpyHostToDevice)); //copy all elements to device after leaving pad number of elements
  CHECK_ERROR(hipMemset(B[0], 0, pad_num_data)); 
  
  int num_ele_this = padded_num_data;
  
  int num_threads = 512;
  int num_blocks = (padded_num_data + num_threads - 1) / num_threads;
  
  // Data Flow up
  for(int h = 1 ; h<=least_pow2;h++){
    num_ele_this = num_ele_this/2;
    CHECK_ERROR(hipMalloc(&this_addr, long(num_ele_this) * long(sizeof(uint32_t))));
    B[h]=this_addr;
    prefix_up<<<num_blocks, num_threads>>>(B[h], B[h-1],num_ele_this);
  }  
  
  CHECK_ERROR(hipDeviceSynchronize());
  // Data Flow Down
  
  num_ele_this=1;
  CHECK_ERROR(hipMalloc(&this_addr, long(num_ele_this) * long(sizeof(uint32_t))));
  C[least_pow2]=this_addr;
  prefix_down<<<num_blocks, num_threads>>>(B[least_pow2],C[least_pow2],nullptr,num_ele_this);
  CHECK_ERROR(hipFree(B[least_pow2]));
  for(int h = least_pow2-1;h>=0;h--){
    num_ele_this*=2;
    CHECK_ERROR(hipMalloc(&this_addr, long(num_ele_this) * long(sizeof(uint32_t))));
    C[h]=this_addr;
    prefix_down<<<num_blocks, num_threads>>>(B[h],C[h],C[h+1],num_ele_this);
    if(h!=0)
      CHECK_ERROR(hipFree(B[h]));
    if(h-2<=least_pow2)
      CHECK_ERROR(hipFree(C[h+2]));
  }
  CHECK_ERROR(hipFree(C[1]));
  CHECK_ERROR(hipDeviceSynchronize());
  
  // free temp arrays
  // for(int h = 1 ; h<=least_pow2;h++){
  //   CHECK_ERROR(hipFree(B[h]));
  //   CHECK_ERROR(hipFree(C[h]));
  // }
  uint32_t *d_prefix_arr = C[0];
  uint32_t *d_data_arr = B[0];
  CHECK_ERROR(hipDeviceSynchronize());
  // Sorting 

  for (int64_t step = 2; step <= padded_num_data; step <<= 1) {
    for (int64_t substep = step >> 1; substep > 0; substep >>= 1) {
      CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_step), &step, sizeof(int64_t)));
      CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_substep), &substep, sizeof(int64_t)));
      sortcu<<<num_blocks, num_threads>>>(d_data_arr,d_prefix_arr, padded_num_data);
  
    }
  }
  
  CHECK_ERROR(hipDeviceSynchronize());
  CHECK_ERROR(hipMemcpy(h_data, d_data_arr+pad_num_data, num_data*sizeof(uint32_t),
                        hipMemcpyDeviceToHost));
   // #ifdef DEBUG
  uint32_t *prefix_arr= new uint32_t[num_data];
    CHECK_ERROR(hipMemcpy(prefix_arr, d_prefix_arr+pad_num_data, num_data*sizeof(uint32_t),
                        hipMemcpyDeviceToHost));
    for(int i =0;i<num_data;i++){
      if(i>0 && prefix_arr[i]==prefix_arr[i-1]){
            continue;
        }
        if(i<num_data-1  && prefix_arr[i]==prefix_arr[i+1]){
            continue;
        }
      cout<<prefix_arr[i]<<"\t";
      if(i%5==0){
        cout<<"\n";
      }
    }

  
  // #endif

  CHECK_ERROR(hipFree(B[0]));
  CHECK_ERROR(hipFree(C[0]));

  delete[] C;
  delete[] B;
  
}

// kernel ran on the GPU
__global__ void d_idx_convert(uint32_t *index_arr,uint32_t * d_arr ,int num_ele_this) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<num_ele_this){
    index_arr[i]=d_arr  [index_arr[i]];
  }
  
}

// kernel ran on the GPU
__global__ void init_idx_arr(uint32_t *index_arr,int num_ele_this) {
  uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<num_ele_this){
    index_arr[i]=i;
  }
  
}

// kernel ran on the GPU
__global__ void prefix_down(uint32_t *B_h, uint32_t *C_h, uint32_t *C_hp1, int num_ele_this) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i==0)
    C_h[0]=B_h[0];
  else if(i<num_ele_this && i%2==1){
    C_h[i]=C_hp1[i/2];
  }
  else if(i<num_ele_this && i%2==0){
    C_h[i]=MOD_MAX_SUM_MY(C_hp1[i/2-1],B_h[i]);
  }
}

// kernel ran on the GPU
__global__ void prefix_up(uint32_t *B_this, uint32_t *B_prev, int num_ele_this) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<num_ele_this){
    B_this[i]= MOD_MAX_SUM_MY(B_prev[2*i],B_prev[2*i+1] ) ;
  }
} 

// kernel ran on the GPU
__global__ void sortcu(uint32_t *d_data_arr, uint32_t *d_prefix_arr, int ndata) {
  int64_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<ndata){
    int64_t j = i ^ d_substep;

    uint32_t data_i = d_prefix_arr[i];
    uint32_t data_j = d_prefix_arr[j];
    uint32_t vdata_i = d_data_arr[i];
    uint32_t vdata_j = d_data_arr[j];
    // uint32_t temp_my = 0;
    if (j > i) {
      if ((((d_step & i) == 0) && (data_i > data_j)) ||
          (((d_step & i) != 0) && (data_i < data_j))) {
        d_prefix_arr[i] = data_j;
        d_prefix_arr[j] = data_i;
        d_data_arr[i]=vdata_j;
        d_data_arr[j]=vdata_i;
      }
    }
  }
}

// // kernel ran on the GPU
// __global__ void sortcu(uint32_t *d_data_arr, uint32_t *d_prefix_arr, int ndata) {
//   int i = blockDim.x * blockIdx.x + threadIdx.x;
//   int j = i ^ d_substep;

//   uint32_t data_i = d_prefix_arr[i];
//   uint32_t data_j = d_prefix_arr[j];
//   uint32_t temp_my = 0;
//   if (j > i) {
//     if ((((d_step & i) == 0) && (data_i > data_j)) ||
//         (((d_step & i) != 0) && (data_i < data_j))) {
//       d_prefix_arr[i] = data_j;
//       d_prefix_arr[j] = data_i;
//       temp_my = d_data_arr[i];
//       d_data_arr[i] = d_data_arr[j];
//       d_data_arr[j]=temp_my;

//     }
//   }
// }


// // kernel ran on the GPU
// __global__ void sortcu1(uint32_t *idx_arr, uint32_t *prefix_arr, int ndata) {
//   int i = blockDim.x * blockIdx.x + threadIdx.x;
//   int j = i ^ d_substep;
 
//   uint32_t data_i = prefix_arr[idx_arr[i]];
//   uint32_t data_j = prefix_arr[idx_arr[j]];
//   uint32_t data_i_i = idx_arr[i];
//   uint32_t data_j_i = idx_arr[j];

//   if (j > i) {
//     if ((((d_step & i) == 0) && (data_i > data_j)) ||
//         (((d_step & i) != 0) && (data_i < data_j))) {
//       idx_arr[i] = data_i_i;
//       idx_arr[j] = data_j_i;
//     }
//   }
// }

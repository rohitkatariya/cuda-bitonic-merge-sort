#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>

#include "hip/hip_runtime.h"
#include ""

#include "sortcu.h"

__global__ void sortcu(uint32_t *data, int ndata);

__device__ int d_step;
__device__ int d_substep;
using namespace std;
// check cuda error
void check_cuda_error(hipError_t err, int line) {
  if (err != hipError_t::hipSuccess) {
    std::cerr << "CUDA error at line " << line << " : "
              << hipGetErrorString(err) << "\n";
    std::exit(1);
  }
}

#define CHECK_ERROR(err) check_cuda_error(err, __LINE__)


// sets up and calls GPU kernel
void sort(uint32_t *data, int ndata) {
  uint32_t *h_data = data; //host pointer
  const int num_data = ndata;
  const int num_bytes = num_data * sizeof(uint32_t);

  int least_pow2 = 0;
  while ((1ULL << least_pow2) < num_data) {
    least_pow2++;
  }
  // printf("\nnum_data:%d,lp:%d\n",num_data,least_pow2);
  
  const int padded_num_data = (1 << least_pow2); //number of elements after padding
  const int padded_num_bytes = padded_num_data * sizeof(uint32_t); // number of bytes with padding
  const int pad_num_data = padded_num_data - num_data; // number of elements to be padded

  uint32_t *d_data = nullptr; //device pointer
  CHECK_ERROR(hipMalloc(&d_data, padded_num_bytes)); //allocate padded_num_bytes on device(GPU)
  CHECK_ERROR(hipMemcpy(d_data + pad_num_data, h_data, num_bytes,
                    hipMemcpyHostToDevice)); //copy all elements to device after leaving pad number of elements
  CHECK_ERROR(hipMemset(d_data, 0, pad_num_data)); // set all elements to be padded as 0
  
  // uint32_t *d_prefx_data = nullptr; //device pointer
  // CHECK_ERROR(hipMalloc(&d_prefx_data, padded_num_bytes)); //allocate padded_num_bytes on device(GPU)
  // CHECK_ERROR(hipMemcpy(d_prefx_data + pad_num_data, h_data, num_bytes,
  //                   hipMemcpyHostToDevice));
  // CHECK_ERROR(hipMemset(d_prefx_data, 0, pad_num_data));

  
  uint32_t **B = new uint32_t*[least_pow2+1];
  uint32_t **C = new uint32_t*[least_pow2+1];
  uint32_t *this_addr=nullptr;
  
  CHECK_ERROR(hipMalloc(&this_addr, padded_num_bytes));
  B[0]=this_addr;
  CHECK_ERROR(hipMalloc(&this_addr, padded_num_bytes));
  C[0]=this_addr;


  CHECK_ERROR(hipMemcpy(B[0] + pad_num_data, h_data, num_bytes,
                    hipMemcpyHostToDevice)); //copy all elements to device after leaving pad number of elements
  CHECK_ERROR(hipMemset(B[0], 0, pad_num_data)); 
  CHECK_ERROR(hipMemcpy(C[0] + pad_num_data, h_data, num_bytes,
                    hipMemcpyHostToDevice)); //copy all elements to device after leaving pad number of elements
  CHECK_ERROR(hipMemset(C[0], 0, pad_num_data)); 
  int num_ele_this = padded_num_bytes;

  // Data Flow up
  for(int h = 1 ; h<=least_pow2;h++){
    num_ele_this = num_ele_this/2;
    printf("\nallocating %d",num_ele_this/ sizeof(uint32_t));
    CHECK_ERROR(hipMalloc(&this_addr, num_ele_this));
    B[h]=this_addr;
    CHECK_ERROR(hipMalloc(&this_addr, num_ele_this));
    C[h]=this_addr;
  }
  // CHECK_ERROR(hipMemset(B[0], 0, pad_num_data));
  // int num_threads = 512;
  // int num_blocks = (padded_num_data + num_threads - 1) / num_threads;
  
  // for(int i =0;i<padded_num_bytes;i++){
  //   B[0][i]=d_data[i];
  // }
  return ;
  
  
  // for (int step = 2; step <= padded_num_data; step <<= 1) {
  //   for (int substep = step >> 1; substep > 0; substep >>= 1) {
  //     CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_step), &step, sizeof(int)));
  //     CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_substep), &substep, sizeof(int)));
  //     sortcu<<<num_blocks, num_threads>>>(d_data, padded_num_data);
  //   }
  // }

  // CHECK_ERROR(hipDeviceSynchronize());

  // CHECK_ERROR(hipMemcpy(h_data, d_data + pad_num_data, num_bytes,
  //                   hipMemcpyDeviceToHost));

  // CHECK_ERROR(hipFree(d_data));
}


// kernel ran on the GPU
__global__ void sortcu(uint32_t *B_this, int B_prev) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = i ^ d_substep;
} 

// kernel ran on the GPU
__global__ void sortcu(uint32_t *data, int ndata) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = i ^ d_substep;
 
  uint32_t data_i = data[i];
  uint32_t data_j = data[j];

  if (j > i) {
    if ((((d_step & i) == 0) && (data_i > data_j)) ||
        (((d_step & i) != 0) && (data_i < data_j))) {
      data[i] = data_j;
      data[j] = data_i;
    }
  }
}

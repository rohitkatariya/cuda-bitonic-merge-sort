#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>

#include "hip/hip_runtime.h"
#include ""

#include "sortcu.h"

__global__ void sortcu(uint32_t *data, int ndata);

__device__ int d_step;
__device__ int d_substep;

// check cuda error
void check_cuda_error(hipError_t err, int line) {
  if (err != hipError_t::hipSuccess) {
    std::cerr << "CUDA error at line " << line << " : "
              << hipGetErrorString(err) << "\n";
    std::exit(1);
  }
}

#define CHECK_ERROR(err) check_cuda_error(err, __LINE__)

// sets up and calls GPU kernel
void sort(uint32_t *data, int ndata) {
  uint32_t *h_data = data;
  const int num_data = ndata;
  const int num_bytes = num_data * sizeof(uint32_t);

  int least_pow2 = 0;
  while ((1ULL << least_pow2) < num_data) {
    least_pow2++;
  }

  const int padded_num_data = (1 << least_pow2);
  const int padded_num_bytes = padded_num_data * sizeof(uint32_t);
  const int pad_num_data = padded_num_data - num_data;

  uint32_t *d_data = nullptr;
  CHECK_ERROR(hipMalloc(&d_data, padded_num_bytes));
  CHECK_ERROR(hipMemcpy(d_data + pad_num_data, h_data, num_bytes,
                    hipMemcpyHostToDevice));
  CHECK_ERROR(hipMemset(d_data, 0, pad_num_data));

  int num_threads = 512;
  int num_blocks = (padded_num_data + num_threads - 1) / num_threads;

  for (int step = 2; step <= padded_num_data; step <<= 1) {
    for (int substep = step >> 1; substep > 0; substep >>= 1) {
      CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_step), &step, sizeof(int)));
      CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_substep), &substep, sizeof(int)));
      sortcu<<<num_blocks, num_threads>>>(d_data, padded_num_data);
    }
  }

  CHECK_ERROR(hipDeviceSynchronize());

  CHECK_ERROR(hipMemcpy(h_data, d_data + pad_num_data, num_bytes,
                    hipMemcpyDeviceToHost));

  CHECK_ERROR(hipFree(d_data));
}

// kernel ran on the GPU
__global__ void sortcu(uint32_t *data, int ndata) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = i ^ d_substep;

  uint32_t data_i = data[i];
  uint32_t data_j = data[j];

  if (j > i) {
    if ((((d_step & i) == 0) && (data_i > data_j)) ||
        (((d_step & i) != 0) && (data_i < data_j))) {
      data[i] = data_j;
      data[j] = data_i;
    }
  }
}
